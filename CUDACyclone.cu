#include "hip/hip_runtime.h"
#include <cstdint>
#include <hip/hip_runtime.h>
#include <>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <iostream>
#include <iomanip>
#include <sstream>
#include <string>
#include <thread>
#include <chrono>
#include <cmath>
#include <csignal>
#include <atomic>

#include "CUDAMath.h"
#include "CUDAStructures.h"

// Verify unsigned long long size
static_assert(sizeof(unsigned long long) == 8, "unsigned long long must be 64 bits");

// Local FoundResult struct for both host and device
struct FoundResult {
    int threadId;
    int iter;
    unsigned long long scalar_val[4];
    unsigned long long Rx_val[4];
    unsigned long long Ry_val[4];
};

// Declarations for functions defined in CUDAUtils.cu
__device__ unsigned long long warp_reduce_add_ull(unsigned long long val);
__device__ bool hash160_prefix_equals(const uint8_t h20[20], uint32_t target_prefix);
__device__ bool hash160_matches_prefix_then_full(const uint8_t h20[20], const uint8_t target[20], uint32_t target_prefix);
__device__ void sub256_u64_inplace(unsigned long long a[4], unsigned long long b);
__device__ void inc256_device(unsigned long long a[4], unsigned long long b);
__host__ bool hexToLE64(const std::string& hex, unsigned long long out[4]);
__host__ void sub256(const unsigned long long a[4], const unsigned long long b[4], unsigned long long out[4]);
__host__ void add256_u64(const unsigned long long a[4], unsigned long long b, unsigned long long out[4]);
__host__ bool decode_p2pkh_address(const std::string& address, uint8_t hash160[20]);
__host__ long double ld_from_u256(const unsigned long long a[4]);

// Declaration for getHash160_33_from_limbs (assumed defined in CUDAHash.cu)
__device__ void getHash160_33_from_limbs(uint8_t prefix, const unsigned long long x[4], uint8_t h20[20]);

// Namespace for utility functions
namespace CryptoUtils {
    std::string formatHex256(const unsigned long long* limbs) {
        std::ostringstream oss;
        oss << std::hex << std::uppercase << std::setfill('0');
        for (int i = 3; i >= 0; --i) {
            oss << std::setw(16) << limbs[i];
        }
        return oss.str();
    }

    std::string formatCompressedPubHex(const unsigned long long* Rx, const unsigned long long* Ry) {
        uint8_t out[33];
        out[0] = (Ry[0] & 1ULL) ? 0x03 : 0x02;
        int off = 1;
        for (int limb = 3; limb >= 0; --limb) {
            unsigned long long v = Rx[limb];
            out[off+0] = (uint8_t)(v >> 56); out[off+1] = (uint8_t)(v >> 48);
            out[off+2] = (uint8_t)(v >> 40); out[off+3] = (uint8_t)(v >> 32);
            out[off+4] = (uint8_t)(v >> 24); out[off+5] = (uint8_t)(v >> 16);
            out[off+6] = (uint8_t)(v >> 8);  out[off+7] = (uint8_t)(v >> 0);
            off += 8;
        }
        static const char* hexd = "0123456789ABCDEF";
        std::string s;
        s.resize(66);
        for (int i = 0; i < 33; ++i) {
            s[2*i] = hexd[(out[i] >> 4) & 0xF];
            s[2*i+1] = hexd[out[i] & 0xF];
        }
        return s;
    }
}

static volatile sig_atomic_t g_sigint = 0;
static void handle_sigint(int) { g_sigint = 1; }

__device__ __forceinline__ int load_found_flag_relaxed(const int* p) {
    return *((const volatile int*)p);
}

__device__ __forceinline__ bool warp_found_ready(const int* __restrict__ d_found_flag, unsigned full_mask, unsigned lane) {
    int f = 0;
    if (lane == 0) f = load_found_flag_relaxed(d_found_flag);
    f = __shfl_sync(full_mask, f, 0);
    return f == FOUND_READY;
}

// Test kernel to verify c_Gx and c_Gy
__global__ void test_constant_memory(unsigned long long* out, int batch_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= batch_size / 2) return;
    for (int i = 0; i < 4; ++i) {
        out[idx * 8 + i] = c_Gx[idx * 4 + i];
        out[idx * 8 + i + 4] = c_Gy[idx * 4 + i];
    }
    if (idx == 0 && threadIdx.x == 0 && blockIdx.x == 0) {
        printf("test_constant_memory: c_Gx[0]=%llx:%llx:%llx:%llx, c_Gy[0]=%llx:%llx:%llx:%llx\n",
               c_Gx[0], c_Gx[1], c_Gx[2], c_Gx[3], c_Gy[0], c_Gy[1], c_Gy[2], c_Gy[3]);
    }
}

// Test kernel to verify precomputed tables
__global__ void test_precomputed_tables(unsigned long long* d_pre_Gx, unsigned long long* d_pre_Gy, unsigned long long* out, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size) return;
    for (int i = 0; i < 4; ++i) {
        out[idx * 8 + i] = d_pre_Gx[idx * 4 + i];
        out[idx * 8 + i + 4] = d_pre_Gy[idx * 4 + i];
    }
    if (idx <= 1 && threadIdx.x == 0 && blockIdx.x == 0) {
        printf("test_precomputed_tables: idx=%d, pre_x=%llx:%llx:%llx:%llx, pre_y=%llx:%llx:%llx:%llx\n",
               idx, out[idx*8], out[idx*8+1], out[idx*8+2], out[idx*8+3],
               out[idx*8+4], out[idx*8+5], out[idx*8+6], out[idx*8+7]);
    }
}

// Modified scalarMulKernelBase for debugging
__global__ void scalarMulKernelBase(
    const unsigned long long* __restrict__ scalars,
    unsigned long long* __restrict__ outX,
    unsigned long long* __restrict__ outY,
    unsigned long long threadsTotal,
    const unsigned long long* __restrict__ d_pre_Gx,
    const unsigned long long* __restrict__ d_pre_Gy,
    const unsigned long long* __restrict__ d_pre_phiGx,
    const unsigned long long* __restrict__ d_pre_phiGy
) {
    unsigned long long gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= threadsTotal) return;

    unsigned long long k1[4], k2[4], scalar[4];
    #pragma unroll
    for (int i = 0; i < 4; ++i) {
        scalar[i] = scalars[gid * 4 + i];
    }
    split_glv(scalar, k1, k2);

    if (threadIdx.x == 0 && blockIdx.x == 0) {
        printf("scalarMulKernelBase: gid=%llu, scalar=%llx:%llx:%llx:%llx, k1=%llx:%llx:%llx:%llx, k2=%llx:%llx:%llx:%llx\n",
               gid, scalar[0], scalar[1], scalar[2], scalar[3],
               k1[0], k1[1], k1[2], k1[3], k2[0], k2[1], k2[2], k2[3]);
    }

    JacobianPoint R;
    if (k1[3] == 0 && k1[2] == 0 && k1[1] == 0 && k1[0] <= 0xFFFFFFFF) {
        pointSetG(R);
        for (uint64_t i = 0; i < k1[0]; ++i) {
            pointDoubleJacobian(R, R);
            if (threadIdx.x == 0 && blockIdx.x == 0 && i % 10 == 0) {
                printf("scalarMulKernelBase: doubling %llu, R.x=%llx:%llx:%llx:%llx, R.z=%llx:%llx:%llx:%llx, R.infinity=%d\n",
                       i, R.x[0], R.x[1], R.x[2], R.x[3], R.z[0], R.z[1], R.z[2], R.z[3], R.infinity);
            }
        }
        pointToAffine(R, outX + gid * 4, outY + gid * 4);
    } else {
        pointSetInfinity(R);
        int msb_k1 = find_msb(k1);
        int msb_k2 = find_msb(k2);
        int max_msb = (msb_k1 > msb_k2) ? msb_k1 : msb_k2;
        for (int i = max_msb; i >= 0; i -= PRECOMPUTE_WINDOW) {
            for (int j = 0; j < PRECOMPUTE_WINDOW && i - j >= 0; ++j) {
                pointDoubleJacobian(R, R);
            }
            int pos = i - PRECOMPUTE_WINDOW + 1;
            if (pos < 0) pos = 0;
            uint32_t window_k1 = get_window(k1, pos);
            if (window_k1) {
                unsigned long long Qx[4], Qy[4];
                fieldCopy(d_pre_Gx + (window_k1 - 1) * 4, Qx);
                fieldCopy(d_pre_Gy + (window_k1 - 1) * 4, Qy);
                pointAddMixed(R, Qx, Qy, false, R);
            }
            uint32_t window_k2 = get_window(k2, pos);
            if (window_k2) {
                unsigned long long Qx[4], Qy[4];
                fieldCopy(d_pre_phiGx + (window_k2 - 1) * 4, Qx);
                fieldCopy(d_pre_phiGy + (window_k2 - 1) * 4, Qy);
                pointAddMixed(R, Qx, Qy, false, R);
            }
        }
        pointToAffine(R, outX + gid * 4, outY + gid * 4);
    }

    if (threadIdx.x == 0 && blockIdx.x == 0) {
        printf("scalarMulKernelBase: gid=%llu, outX=%llx:%llx:%llx:%llx, outY=%llx:%llx:%llx:%llx\n",
               gid, outX[gid*4], outX[gid*4+1], outX[gid*4+2], outX[gid*4+3],
               outY[gid*4], outY[gid*4+1], outY[gid*4+2], outY[gid*4+3]);
    }
}

__launch_bounds__(256, 2)
__global__ void fused_ec_hash(
    JacobianPoint* __restrict__ P,
    JacobianPoint* __restrict__ R,
    unsigned long long* __restrict__ start_scalars,
    unsigned long long* __restrict__ counts256,
    unsigned long long threadsTotal,
    uint32_t batch_size,
    uint32_t max_batches_per_launch,
    int* __restrict__ d_found_flag,
    FoundResult* __restrict__ d_found_result,
    unsigned long long* __restrict__ hashes_accum,
    unsigned int* __restrict__ d_any_left
) {
    const int B = (int)batch_size;
    if (B <= 0 || (B & 1) || B > MAX_BATCH_SIZE) return;
    const int half = B >> 1;

    const unsigned long long gid = (unsigned long long)blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= threadsTotal) return;

    const unsigned lane = (unsigned)(threadIdx.x & (WARP_SIZE - 1));
    const unsigned full_mask = 0xFFFFFFFFu;
    if (warp_found_ready(d_found_flag, full_mask, lane)) return;

    unsigned int local_hashes = 0;
    #define FLUSH_THRESHOLD 65536u
    #define WARP_FLUSH_HASHES() do { \
        unsigned long long v = warp_reduce_add_ull((unsigned long long)local_hashes); \
        if (lane == 0 && v) atomicAdd(hashes_accum, v); \
        local_hashes = 0; \
    } while (0)
    #define MAYBE_WARP_FLUSH() do { if ((local_hashes & (FLUSH_THRESHOLD - 1u)) == 0u) WARP_FLUSH_HASHES(); } while (0)

    JacobianPoint P_local = P[gid];
    if (lane == 0 && blockIdx.x == 0) {
        printf("fused_ec_hash: gid=%llu, P_local.x=%llx:%llx:%llx:%llx, P_local.y=%llx:%llx:%llx:%llx, P_local.z=%llx:%llx:%llx:%llx, P_local.infinity=%d\n",
               gid, P_local.x[0], P_local.x[1], P_local.x[2], P_local.x[3],
               P_local.y[0], P_local.y[1], P_local.y[2], P_local.y[3],
               P_local.z[0], P_local.z[1], P_local.z[2], P_local.z[3], P_local.infinity);
    }
    unsigned long long S[4], rem[4];
    #pragma unroll
    for (int i = 0; i < 4; ++i) {
        S[i] = start_scalars[gid*4 + i];
        rem[i] = counts256[gid*4 + i];
    }

    if (isZero256(rem)) {
        R[gid] = P_local;
        WARP_FLUSH_HASHES();
        return;
    }

    uint32_t batches_done = 0;
    extern __shared__ unsigned long long shared_mem[];
    unsigned long long* z_values = shared_mem;

    while (batches_done < max_batches_per_launch && ge256_u64(rem, (unsigned long long)B)) {
        if (warp_found_ready(d_found_flag, full_mask, lane)) {
            WARP_FLUSH_HASHES();
            return;
        }

        // Batch point additions
        for (int i = 0; i < half; ++i) {
            if (lane + i * WARP_SIZE >= batch_size / 2) continue;
            JacobianPoint Q;
            if ((lane + i * WARP_SIZE) * 4 < batch_size * 4) {
                fieldCopy(c_Gx + (lane + i * WARP_SIZE) * 4, Q.x);
                fieldCopy(c_Gy + (lane + i * WARP_SIZE) * 4, Q.y);
            } else {
                fieldSetZero(Q.x);
                fieldSetZero(Q.y);
            }
            fieldSetOne(Q.z);
            Q.infinity = false;
            pointAddMixed(P_local, Q.x, Q.y, Q.infinity, P_local);
            if (lane + half + i * WARP_SIZE >= batch_size / 2) continue;
            if ((lane + half + i * WARP_SIZE) * 4 < batch_size * 4) {
                fieldCopy(c_Gx + (lane + half + i * WARP_SIZE) * 4, Q.x);
                fieldCopy(c_Gy + (lane + half + i * WARP_SIZE) * 4, Q.y);
            } else {
                fieldSetZero(Q.x);
                fieldSetZero(Q.y);
            }
            pointAddMixed(P_local, Q.x, Q.y, Q.infinity, P_local);
            if (lane == 0 && blockIdx.x == 0) {
                printf("fused_ec_hash: Block %d, after pointAddMixed, P_local.x=%llx:%llx:%llx:%llx, P_local.z=%llx:%llx:%llx:%llx, P_local.infinity=%d\n",
                       blockIdx.x, P_local.x[0], P_local.x[1], P_local.x[2], P_local.x[3],
                       P_local.z[0], P_local.z[1], P_local.z[2], P_local.z[3], P_local.infinity);
            }
        }
        __syncthreads(); // Ensure all point additions are complete

        // Batch inversion
        bool skip_inversion = P_local.infinity || isZero256(P_local.z);
        int any_non_infinity = __syncthreads_count(!skip_inversion);
        if (lane < B && !skip_inversion) {
            fieldCopy(P_local.z, z_values + lane * 4);
            if (lane == 0 && blockIdx.x == 0) {
                printf("fused_ec_hash: Block %d, lane %d, writing z_values[0]=%llx:%llx:%llx:%llx\n",
                       blockIdx.x, lane, z_values[0], z_values[1], z_values[2], z_values[3]);
            }
        } else if (lane < B) {
            fieldSetZero(z_values + lane * 4);
            if (lane == 0 && blockIdx.x == 0) {
                printf("fused_ec_hash: Block %d, lane %d, setting z_values[0]=0:0:0:0 (skip_inversion=%d)\n",
                       blockIdx.x, lane, skip_inversion);
            }
        }
        __syncthreads();
        if (lane == 0 && any_non_infinity) {
            batch_modinv_fermat(z_values, z_values, B);
            printf("fused_ec_hash: Block %d, lane 0, after batch_modinv_fermat, z_values[0]=%llx:%llx:%llx:%llx\n",
                   blockIdx.x, z_values[0], z_values[1], z_values[2], z_values[3]);
        }
        __syncthreads();

        // Convert to affine and hash
        unsigned long long x_affine[4], y_affine[4];
        if (lane < B && !skip_inversion) {
            unsigned long long zinv[4], zinv2[4];
            fieldCopy(z_values + lane * 4, zinv);
            fieldSqr_opt_device(zinv, zinv2);
            fieldMul_opt_device(P_local.x, zinv2, x_affine);
            fieldMul_opt_device(zinv, zinv2, zinv2);
            fieldMul_opt_device(P_local.y, zinv2, y_affine);
            if (lane == 0 && blockIdx.x == 0) {
                printf("fused_ec_hash: Block %d, lane %d, x_affine=%llx:%llx:%llx:%llx, y_affine=%llx:%llx:%llx:%llx\n",
                       blockIdx.x, lane, x_affine[0], x_affine[1], x_affine[2], x_affine[3],
                       y_affine[0], y_affine[1], y_affine[2], y_affine[3]);
            }
        } else {
            fieldSetZero(x_affine);
            fieldSetZero(y_affine);
        }

        uint8_t h20[20];
        uint8_t prefix = (y_affine[0] & 1ULL) ? 0x03 : 0x02;
        getHash160_33_from_limbs(prefix, x_affine, h20);
        ++local_hashes;
        MAYBE_WARP_FLUSH();

        bool pref = hash160_prefix_equals(h20, c_target_prefix);
        if (__any_sync(full_mask, pref)) {
            if (pref && hash160_matches_prefix_then_full(h20, c_target_hash160, c_target_prefix)) {
                if (atomicCAS(d_found_flag, FOUND_NONE, FOUND_LOCK) == FOUND_NONE) {
                    d_found_result->threadId = (int)gid;
                    d_found_result->iter = batches_done;
                    #pragma unroll
                    for (int i = 0; i < 4; ++i) {
                        d_found_result->scalar_val[i] = S[i];
                        d_found_result->Rx_val[i] = x_affine[i];
                        d_found_result->Ry_val[i] = y_affine[i];
                    }
                    atomicExch(d_found_flag, FOUND_READY);
                }
            }
        }

        sub256_u64_inplace(rem, (unsigned long long)B);
        inc256_device(S, (unsigned long long)B);
        batches_done++;
        if (isZero256(rem)) {
            atomicOr(d_any_left, 0u);
        } else {
            atomicOr(d_any_left, 1u);
        }
    }

    R[gid] = P_local;
    WARP_FLUSH_HASHES();
}

__global__ void precompute_batch_points_kernel(unsigned long long* d_Gx, unsigned long long* d_Gy, int batch_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= batch_size / 2) return;

    JacobianPoint G, tmp;
    fieldCopy(Gx_d, G.x);
    fieldCopy(Gy_d, G.y);
    fieldSetOne(G.z);
    G.infinity = false;

    // Compute 2^i * G for first half
    for (int i = 0; i < idx; ++i) {
        pointDoubleJacobian(G, tmp);
        G = tmp;
    }
    fieldCopy(G.x, d_Gx + idx * 4);
    fieldCopy(G.y, d_Gy + idx * 4);

    // Compute 2^(i + batch_size/2) * G for second half
    if (idx + batch_size / 2 < batch_size) {
        pointDoubleJacobian(G, tmp);
        G = tmp;
        fieldCopy(G.x, d_Gx + (idx + batch_size / 2) * 4);
        fieldCopy(G.y, d_Gy + (idx + batch_size / 2) * 4);
    }
}

__global__ void compute_phi_base_kernel(const unsigned long long* beta, const unsigned long long* Gx_d, unsigned long long* phi_base_x) {
    fieldMul_opt_device(beta, Gx_d, phi_base_x);
}

std::string human_bytes(size_t bytes) {
    const char* units[] = {"B", "KB", "MB", "GB", "TB"};
    int unit_idx = 0;
    double size = static_cast<double>(bytes);
    while (size >= 1024 && unit_idx < 4) {
        size /= 1024;
        unit_idx++;
    }
    std::stringstream ss;
    ss << std::fixed << std::setprecision(2) << size << " " << units[unit_idx];
    return ss.str();
}

void precompute_g_table_gpu(JacobianPoint base, JacobianPoint phi_base, unsigned long long** d_pre_Gx, unsigned long long** d_pre_Gy, unsigned long long** d_pre_phiGx, unsigned long long** d_pre_phiGy) {
    size_t table_size = PRECOMPUTE_SIZE * 4 * sizeof(unsigned long long);
    size_t total_size = table_size * 4; // 4 tables: Gx, Gy, phiGx, phiGy
    size_t free_mem, total_mem;
    CUDA_CHECK(hipMemGetInfo(&free_mem, &total_mem));
    if (free_mem < total_size + 1e9) { // Reserve ~1GB for other allocations
        std::cerr << "Insufficient VRAM for 2^" << PRECOMPUTE_WINDOW << " precomputed tables (~" << human_bytes(total_size) << ")\n";
        exit(EXIT_FAILURE);
    }

    CUDA_CHECK(hipMalloc(d_pre_Gx, table_size));
    CUDA_CHECK(hipMalloc(d_pre_Gy, table_size));
    CUDA_CHECK(hipMalloc(d_pre_phiGx, table_size));
    CUDA_CHECK(hipMalloc(d_pre_phiGy, table_size));

    int threads = 256;
    int blocks = (PRECOMPUTE_SIZE + threads - 1) / threads;
    printf("precompute_g_table_gpu: base.x=%llx:%llx:%llx:%llx, base.y=%llx:%llx:%llx:%llx, base.infinity=%d\n",
           base.x[0], base.x[1], base.x[2], base.x[3], base.y[0], base.y[1], base.y[2], base.y[3], base.infinity);
    printf("precompute_g_table_gpu: phi_base.x=%llx:%llx:%llx:%llx, phi_base.y=%llx:%llx:%llx:%llx, phi_base.infinity=%d\n",
           phi_base.x[0], phi_base.x[1], phi_base.x[2], phi_base.x[3], phi_base.y[0], phi_base.y[1], phi_base.y[2], phi_base.y[3], phi_base.infinity);
    precompute_table_kernel<<<blocks, threads>>>(base, *d_pre_Gx, *d_pre_Gy, PRECOMPUTE_SIZE);
    CUDA_CHECK(hipDeviceSynchronize());
    std::cout << "precompute_table_kernel (Gx, Gy) completed" << std::endl;
    precompute_table_kernel<<<blocks, threads>>>(phi_base, *d_pre_phiGx, *d_pre_phiGy, PRECOMPUTE_SIZE);
    CUDA_CHECK(hipDeviceSynchronize());
    std::cout << "precompute_table_kernel (phiGx, phiGy) completed" << std::endl;
}

void print_gpu_info(const hipDeviceProp_t& prop, int blocks, int threadsPerBlock, int batch_size, unsigned long long threadsTotal) {
    size_t table_size = PRECOMPUTE_SIZE * 4 * sizeof(unsigned long long) * 4; // 4 tables
    size_t mem_used = (threadsTotal * (4 * 3 + 4 + 4) * sizeof(unsigned long long)) + sizeof(FoundResult) +
                      sizeof(int) + sizeof(unsigned long long) + sizeof(unsigned int) + table_size;
    std::cout << "======== PrePhase: GPU Information ====================\n";
    std::cout << "Device               : " << prop.name << " (compute " << prop.major << "." << prop.minor << ")\n";
    std::cout << "SM                   : " << prop.multiProcessorCount << "\n";
    std::cout << "ThreadsPerBlock      : " << threadsPerBlock << "\n";
    std::cout << "Blocks               : " << blocks << "\n";
    std::cout << "Points batch size    : " << batch_size << "\n";
    std::cout << "Batches/SM           : " << (batch_size / prop.multiProcessorCount) << "\n";
    std::cout << "Precomputed tables    : 2^" << PRECOMPUTE_WINDOW << " points (~" << human_bytes(table_size) << ")\n";
    std::cout << "Memory utilization   : " << std::fixed << std::setprecision(1)
              << (mem_used / (double)prop.totalGlobalMem) * 100.0 << "% ("
              << human_bytes(mem_used) << " / " << human_bytes(prop.totalGlobalMem) << ")\n";
    std::cout << "-------------------------------------------------------\n";
    std::cout << "Total threads        : " << threadsTotal << "\n";
}

int main(int argc, char* argv[]) {
    signal(SIGINT, handle_sigint);

    // Argument parsing
    unsigned long long range_start[4] = {0}, range_end[4] = {0}, range_len[4];
    uint8_t target_hash160[20] = {0};
    int blocks = 64, threadsPerBlock = 32; // Default to match --grid 64,32
    int batch_size = 8;
    uint32_t max_batches_per_launch = 64;
    std::string range_str, address_str, grid_str;
    bool verbose = false;

    for (int i = 1; i < argc; ++i) {
        std::string arg = argv[i];
        if (arg == "--range" && i + 1 < argc) {
            range_str = argv[++i];
        } else if (arg == "--address" && i + 1 < argc) {
            address_str = argv[++i];
        } else if (arg == "--grid" && i + 1 < argc) {
            grid_str = argv[++i];
        } else if (arg == "--slices" && i + 1 < argc) {
            max_batches_per_launch = std::atoi(argv[++i]);
        } else if (arg == "--verbose") {
            verbose = true;
        } else {
            std::cerr << "Unknown argument: " << arg << "\n";
            return EXIT_FAILURE;
        }
    }

    if (range_str.empty() || address_str.empty()) {
        std::cerr << "Usage: " << argv[0] << " --range START:END --address ADDRESS [--grid BLOCKS,THREADS] [--slices SLICES] [--verbose]\n";
        return EXIT_FAILURE;
    }

    // Parse range
    auto colon = range_str.find(':');
    if (colon == std::string::npos) {
        std::cerr << "Invalid range format. Use START:END\n";
        return EXIT_FAILURE;
    }
    std::string start_str = range_str.substr(0, colon);
    std::string end_str = range_str.substr(colon + 1);
    if (!hexToLE64(start_str, range_start) || !hexToLE64(end_str, range_end)) {
        std::cerr << "Invalid range hex values\n";
        return EXIT_FAILURE;
    }
    sub256(range_end, range_start, range_len);

    // Parse address
    if (!decode_p2pkh_address(address_str, target_hash160)) {
        std::cerr << "Invalid Bitcoin address\n";
        return EXIT_FAILURE;
    }

    // Parse grid and debug
    if (!grid_str.empty()) {
        auto comma = grid_str.find(',');
        if (comma != std::string::npos) {
            blocks = std::atoi(grid_str.substr(0, comma).c_str());
            threadsPerBlock = std::atoi(grid_str.substr(comma + 1).c_str());
            if (threadsPerBlock % WARP_SIZE != 0 || threadsPerBlock > 1024) {
                std::cerr << "Threads per block must be multiple of " << WARP_SIZE << " and <= 1024\n";
                return EXIT_FAILURE;
            }
        }
    }
    if (verbose) {
        std::cout << "Parsed grid: blocks=" << blocks << ", threadsPerBlock=" << threadsPerBlock << "\n";
    }

    // Validate batch size
    if (batch_size <= 0 || (batch_size & 1) || batch_size > MAX_BATCH_SIZE) {
        std::cerr << "Batch size must be even and <= " << MAX_BATCH_SIZE << "\n";
        return EXIT_FAILURE;
    }

    // Debug constants from CUDAStructures.h
    unsigned long long h_n[4], h_beta[4], h_b1[4], h_b2[4], h_a1[4], h_a2[4], h_p[4], h_mu[5];
    CUDA_CHECK(hipMemcpyFromSymbol(h_n, HIP_SYMBOL(c_n), 4 * sizeof(unsigned long long)));
    CUDA_CHECK(hipMemcpyFromSymbol(h_beta, HIP_SYMBOL(c_beta), 4 * sizeof(unsigned long long)));
    CUDA_CHECK(hipMemcpyFromSymbol(h_b1, HIP_SYMBOL(c_b1), 4 * sizeof(unsigned long long)));
    CUDA_CHECK(hipMemcpyFromSymbol(h_b2, HIP_SYMBOL(c_b2), 4 * sizeof(unsigned long long)));
    CUDA_CHECK(hipMemcpyFromSymbol(h_a1, HIP_SYMBOL(c_a1), 4 * sizeof(unsigned long long)));
    CUDA_CHECK(hipMemcpyFromSymbol(h_a2, HIP_SYMBOL(c_a2), 4 * sizeof(unsigned long long)));
    CUDA_CHECK(hipMemcpyFromSymbol(h_p, HIP_SYMBOL(c_p), 4 * sizeof(unsigned long long)));
    CUDA_CHECK(hipMemcpyFromSymbol(h_mu, HIP_SYMBOL(c_mu), 5 * sizeof(unsigned long long)));
    if (verbose) {
        std::cout << "c_n: " << std::hex << h_n[0] << ":" << h_n[1] << ":" << h_n[2] << ":" << h_n[3] << std::endl;
        std::cout << "c_beta: " << std::hex << h_beta[0] << ":" << h_beta[1] << ":" << h_beta[2] << ":" << h_beta[3] << std::endl;
        std::cout << "c_b1: " << std::hex << h_b1[0] << ":" << h_b1[1] << ":" << h_b1[2] << ":" << h_b1[3] << std::endl;
        std::cout << "c_b2: " << std::hex << h_b2[0] << ":" << h_b2[1] << ":" << h_b2[2] << ":" << h_b2[3] << std::endl;
        std::cout << "c_a1: " << std::hex << h_a1[0] << ":" << h_a1[1] << ":" << h_a1[2] << ":" << h_a1[3] << std::endl;
        std::cout << "c_a2: " << std::hex << h_a2[0] << ":" << h_a2[1] << ":" << h_a2[2] << ":" << h_a2[3] << std::endl;
        std::cout << "c_p: " << std::hex << h_p[0] << ":" << h_p[1] << ":" << h_p[2] << ":" << h_p[3] << std::endl;
        std::cout << "c_mu: " << std::hex << h_mu[0] << ":" << h_mu[1] << ":" << h_mu[2] << ":" << h_mu[3] << ":" << h_mu[4] << std::endl;
    }

    // Initialize Gx_d and Gy_d with correct secp256k1 generator point
    unsigned long long h_Gx_d[4] = {
        0x9f2815b16f81798ULL, 0x29bfcdb2dce28d95ULL, 0x55a06295ce870b07ULL, 0x79be667ef9dcbbacULL
    };
    unsigned long long h_Gy_d[4] = {
        0x9c47d08ffb10d4b8ULL, 0xfd17b448a6855419ULL, 0x5da4fbfc0e1108a8ULL, 0x483ada7726a3c465ULL
    };
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(Gx_d), h_Gx_d, 4 * sizeof(unsigned long long)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(Gy_d), h_Gy_d, 4 * sizeof(unsigned long long)));

    // GPU setup
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
    unsigned long long threadsTotal = (unsigned long long)blocks * threadsPerBlock;
    if (verbose) {
        print_gpu_info(prop, blocks, threadsPerBlock, batch_size, threadsTotal);
    }

    std::cout << "Batch size: " << batch_size << std::endl;

    // Precompute tables
    JacobianPoint h_base, h_phi_base;
    fieldCopy(h_Gx_d, h_base.x);
    fieldCopy(h_Gy_d, h_base.y);
    fieldSetOne(h_base.z);
    h_base.infinity = false;

    // Compute phi_base.x on GPU
    unsigned long long *d_beta, *d_Gx_d, *d_phi_base_x;
    CUDA_CHECK(hipMalloc(&d_beta, 4 * sizeof(unsigned long long)));
    CUDA_CHECK(hipMalloc(&d_Gx_d, 4 * sizeof(unsigned long long)));
    CUDA_CHECK(hipMalloc(&d_phi_base_x, 4 * sizeof(unsigned long long)));
    unsigned long long h_beta_local[4] = {
        0x6B3C4F7EULL, 0x8DE6997DULL, 0x7CF27B18ULL, 0x00000000ULL
    };
    CUDA_CHECK(hipMemcpy(d_beta, h_beta_local, 4 * sizeof(unsigned long long), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_Gx_d, h_Gx_d, 4 * sizeof(unsigned long long), hipMemcpyHostToDevice));
    compute_phi_base_kernel<<<1, 1>>>(d_beta, d_Gx_d, d_phi_base_x);
    CUDA_CHECK(hipDeviceSynchronize());
    std::cout << "compute_phi_base_kernel completed" << std::endl;
    CUDA_CHECK(hipMemcpy(h_phi_base.x, d_phi_base_x, 4 * sizeof(unsigned long long), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_beta));
    CUDA_CHECK(hipFree(d_Gx_d));
    CUDA_CHECK(hipFree(d_phi_base_x));
    fieldCopy(h_Gy_d, h_phi_base.y);
    fieldSetOne(h_phi_base.z);
    h_phi_base.infinity = false;

    // Precompute tables
    unsigned long long *d_pre_Gx_local, *d_pre_Gy_local, *d_pre_phiGx_local, *d_pre_phiGy_local;
    precompute_g_table_gpu(h_base, h_phi_base, &d_pre_Gx_local, &d_pre_Gy_local, &d_pre_phiGx_local, &d_pre_phiGy_local);

    // Test precomputed tables
    unsigned long long *d_test_table_out;
    CUDA_CHECK(hipMalloc(&d_test_table_out, PRECOMPUTE_SIZE * 8 * sizeof(unsigned long long)));
    test_precomputed_tables<<<(PRECOMPUTE_SIZE + 255) / 256, 256>>>(d_pre_Gx_local, d_pre_Gy_local, d_test_table_out, PRECOMPUTE_SIZE);
    CUDA_CHECK(hipDeviceSynchronize());
    std::cout << "test_precomputed_tables completed" << std::endl;
    unsigned long long h_test_table_out[PRECOMPUTE_SIZE * 8];
    CUDA_CHECK(hipMemcpy(h_test_table_out, d_test_table_out, PRECOMPUTE_SIZE * 8 * sizeof(unsigned long long), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_test_table_out));
    if (verbose) {
        for (int i = 0; i < 2 && i < PRECOMPUTE_SIZE; ++i) {
            std::cout << "d_pre_Gx[" << i << "]: " << std::hex << h_test_table_out[i*8] << ":" << h_test_table_out[i*8+1] << ":"
                      << h_test_table_out[i*8+2] << ":" << h_test_table_out[i*8+3] << std::endl;
            std::cout << "d_pre_Gy[" << i << "]: " << std::hex << h_test_table_out[i*8+4] << ":" << h_test_table_out[i*8+5] << ":"
                      << h_test_table_out[i*8+6] << ":" << h_test_table_out[i*8+7] << std::endl;
        }
    }

    // Precompute batch points on GPU
    unsigned long long *d_Gx, *d_Gy;
    CUDA_CHECK(hipMalloc(&d_Gx, batch_size * 4 * sizeof(unsigned long long)));
    CUDA_CHECK(hipMalloc(&d_Gy, batch_size * 4 * sizeof(unsigned long long)));
    int threads = 256;
    int blocks_batch = (batch_size / 2 + threads - 1) / threads;
    precompute_batch_points_kernel<<<blocks_batch, threads>>>(d_Gx, d_Gy, batch_size);
    CUDA_CHECK(hipDeviceSynchronize());
    std::cout << "precompute_batch_points_kernel completed" << std::endl;

    // Test c_Gx and c_Gy
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_Gx), d_Gx, batch_size * 4 * sizeof(unsigned long long)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_Gy), d_Gy, batch_size * 4 * sizeof(unsigned long long)));
    unsigned long long *d_test_out;
    CUDA_CHECK(hipMalloc(&d_test_out, (batch_size / 2) * 8 * sizeof(unsigned long long)));
    test_constant_memory<<<1, threads>>>(d_test_out, batch_size);
    CUDA_CHECK(hipDeviceSynchronize());
    std::cout << "test_constant_memory completed" << std::endl;
    unsigned long long h_test_out[(batch_size / 2) * 8];
    CUDA_CHECK(hipMemcpy(h_test_out, d_test_out, (batch_size / 2) * 8 * sizeof(unsigned long long), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_test_out));
    if (verbose) {
        std::cout << "c_Gx[0]: " << std::hex << h_test_out[0] << ":" << h_test_out[1] << ":"
                  << h_test_out[2] << ":" << h_test_out[3] << std::endl;
        std::cout << "c_Gy[0]: " << std::hex << h_test_out[4] << ":" << h_test_out[5] << ":"
                  << h_test_out[6] << ":" << h_test_out[7] << std::endl;
    }

    CUDA_CHECK(hipFree(d_Gx));
    CUDA_CHECK(hipFree(d_Gy));

    // Set target
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_target_hash160), target_hash160, sizeof(target_hash160)));
    uint32_t target_prefix = *(uint32_t*)target_hash160;
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_target_prefix), &target_prefix, sizeof(target_prefix)));

    // Allocate device memory
    unsigned long long *d_start_scalars, *d_counts256;
    unsigned long long *d_hashes_accum;
    int *d_found_flag;
    unsigned int *d_any_left;
    FoundResult *d_found_result;
    JacobianPoint *d_P, *d_R;
    CUDA_CHECK(hipMalloc(&d_start_scalars, threadsTotal * 4 * sizeof(unsigned long long)));
    CUDA_CHECK(hipMalloc(&d_counts256, threadsTotal * 4 * sizeof(unsigned long long)));
    CUDA_CHECK(hipMalloc(&d_P, threadsTotal * sizeof(JacobianPoint)));
    CUDA_CHECK(hipMalloc(&d_R, threadsTotal * sizeof(JacobianPoint)));
    CUDA_CHECK(hipMalloc(&d_found_flag, sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_found_result, sizeof(FoundResult)));
    CUDA_CHECK(hipMalloc(&d_hashes_accum, sizeof(unsigned long long)));
    CUDA_CHECK(hipMalloc(&d_any_left, sizeof(unsigned int)));

    // Initialize scalars and counts
    unsigned long long *h_start_scalars = nullptr, *h_counts256 = nullptr;
    CUDA_CHECK(hipHostMalloc(&h_start_scalars, threadsTotal * 4 * sizeof(unsigned long long)));
    CUDA_CHECK(hipHostMalloc(&h_counts256, threadsTotal * 4 * sizeof(unsigned long long)));
    for (unsigned long long i = 0; i < threadsTotal; ++i) {
        add256_u64(range_start, i, h_start_scalars + i * 4);
        unsigned long long end_plus_1[4];
        add256_u64(range_start, threadsTotal, end_plus_1);
        unsigned long long count[4];
        sub256(range_end, h_start_scalars + i * 4, count);
        if (ge256_u64(end_plus_1, range_end[0])) {
            unsigned long long remaining[4];
            sub256(end_plus_1, range_end, remaining);
            sub256(count, remaining, count);
        }
        fieldCopy(count, h_counts256 + i * 4);
    }
    if (verbose) {
        std::cout << "First scalar: " << std::hex << h_start_scalars[0] << ":" << h_start_scalars[1] << ":"
                  << h_start_scalars[2] << ":" << h_start_scalars[3] << std::endl;
        std::cout << "First count: " << std::hex << h_counts256[0] << ":" << h_counts256[1] << ":"
                  << h_counts256[2] << ":" << h_counts256[3] << std::endl;
    }
    CUDA_CHECK(hipMemcpy(d_start_scalars, h_start_scalars, threadsTotal * 4 * sizeof(unsigned long long), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_counts256, h_counts256, threadsTotal * 4 * sizeof(unsigned long long), hipMemcpyHostToDevice));

    // Initialize points
    unsigned long long *d_outX, *d_outY;
    CUDA_CHECK(hipMalloc(&d_outX, threadsTotal * 4 * sizeof(unsigned long long)));
    CUDA_CHECK(hipMalloc(&d_outY, threadsTotal * 4 * sizeof(unsigned long long)));
    scalarMulKernelBase<<<blocks, threadsPerBlock>>>(d_start_scalars, d_outX, d_outY, threadsTotal, d_pre_Gx_local, d_pre_Gy_local, d_pre_phiGx_local, d_pre_phiGy_local);
    CUDA_CHECK(hipDeviceSynchronize());
    std::cout << "scalarMulKernelBase completed" << std::endl;
    JacobianPoint *h_P = new JacobianPoint[threadsTotal];
    unsigned long long *h_outX = new unsigned long long[threadsTotal * 4], *h_outY = new unsigned long long[threadsTotal * 4];
    CUDA_CHECK(hipMemcpy(h_outX, d_outX, threadsTotal * 4 * sizeof(unsigned long long), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_outY, d_outY, threadsTotal * 4 * sizeof(unsigned long long), hipMemcpyDeviceToHost));
    for (unsigned long long i = 0; i < threadsTotal; ++i) {
        fieldCopy(h_outX + i * 4, h_P[i].x);
        fieldCopy(h_outY + i * 4, h_P[i].y);
        fieldSetOne(h_P[i].z);
        h_P[i].infinity = isZero256(h_outX + i * 4) && isZero256(h_outY + i * 4);
    }
    if (verbose) {
        std::cout << "d_P[0].x: " << std::hex << h_P[0].x[0] << ":" << h_P[0].x[1] << ":"
                  << h_P[0].x[2] << ":" << h_P[0].x[3] << std::endl;
        std::cout << "d_P[0].y: " << std::hex << h_P[0].y[0] << ":" << h_P[0].y[1] << ":"
                  << h_P[0].y[2] << ":" << h_P[0].y[3] << std::endl;
        std::cout << "d_P[0].infinity: " << h_P[0].infinity << std::endl;
    }
    CUDA_CHECK(hipMemcpy(d_P, h_P, threadsTotal * sizeof(JacobianPoint), hipMemcpyHostToDevice));
    delete[] h_P; delete[] h_outX; delete[] h_outY;
    CUDA_CHECK(hipFree(d_outX));
    CUDA_CHECK(hipFree(d_outY));

    // Initialize device memory
    CUDA_CHECK(hipMemset(d_found_flag, 0, sizeof(int)));
    CUDA_CHECK(hipMemset(d_hashes_accum, 0, sizeof(unsigned long long)));
    CUDA_CHECK(hipMemset(d_any_left, 0, sizeof(unsigned int)));

    hipStream_t streamKernel;
    CUDA_CHECK(hipStreamCreate(&streamKernel));

    bool stop_all = false, completed_all = false;
    unsigned long long lastHashes = 0;
    auto t0 = std::chrono::high_resolution_clock::now();
    auto tLast = t0;

    std::cout << "\n======== Phase-1: BruteForce (sliced) =================\n";

    while (!stop_all) {
        dim3 gridDim(blocks, 1, 1);
        dim3 blockDim(threadsPerBlock, 1, 1);
        size_t sharedMem = (batch_size + 1) * 4 * sizeof(unsigned long long); // Adjusted for n+1 elements
        fused_ec_hash<<<gridDim, blockDim, sharedMem, streamKernel>>>(
            d_P, d_R, d_start_scalars, d_counts256, threadsTotal, batch_size,
            max_batches_per_launch, d_found_flag, d_found_result, d_hashes_accum, d_any_left
        );
        hipError_t launchErr = hipGetLastError();
        if (launchErr != hipSuccess) {
            std::cerr << "\nKernel launch error: " << hipGetErrorString(launchErr) << "\n";
            stop_all = true;
        }

        while (!stop_all) {
            auto now = std::chrono::high_resolution_clock::now();
            double dt = std::chrono::duration<double>(now - tLast).count();
            if (dt >= 1.0) {
                unsigned long long h_hashes = 0;
                CUDA_CHECK(hipMemcpy(&h_hashes, d_hashes_accum, sizeof(unsigned long long), hipMemcpyDeviceToHost));
                double delta = (double)(h_hashes - lastHashes);
                double mkeys = delta / (dt * 1e6);
                double elapsed = std::chrono::duration<double>(now - t0).count();
                long double total_keys = ld_from_u256(range_len);
                long double prog = total_keys > 0.0L ? ((long double)h_hashes / total_keys) * 100.0L : 0.0L;
                if (prog > 100.0L) prog = 100.0L;
                std::cout << "\rTime: " << std::fixed << std::setprecision(1) << elapsed
                          << " s | Speed: " << std::fixed << std::setprecision(1) << mkeys
                          << " Mkeys/s | Count: " << h_hashes
                          << " | Progress: " << std::fixed << std::setprecision(2) << (double)prog << " %";
                std::cout.flush();
                lastHashes = h_hashes;
                tLast = now;
            }

            int host_found = 0;
            CUDA_CHECK(hipMemcpy(&host_found, d_found_flag, sizeof(int), hipMemcpyDeviceToHost));
            if (host_found == FOUND_READY) {
                stop_all = true;
                break;
            }

            hipError_t qs = hipStreamQuery(streamKernel);
            if (qs == hipSuccess) break;
            if (qs != hipErrorNotReady) {
                CUDA_CHECK(hipGetLastError());
                stop_all = true;
                break;
            }

            std::this_thread::sleep_for(std::chrono::milliseconds(10));
        }

        CUDA_CHECK(hipStreamSynchronize(streamKernel));
        std::cout.flush();
        if (stop_all || g_sigint) break;

        unsigned int h_any = 0;
        CUDA_CHECK(hipMemcpy(&h_any, d_any_left, sizeof(unsigned int), hipMemcpyDeviceToHost));
        std::swap(d_P, d_R);
        if (h_any == 0u) {
            completed_all = true;
            break;
        }
    }

    CUDA_CHECK(hipDeviceSynchronize());
    std::cout << "\n";

    int h_found_flag = 0;
    CUDA_CHECK(hipMemcpy(&h_found_flag, d_found_flag, sizeof(int), hipMemcpyDeviceToHost));
    int exit_code = EXIT_SUCCESS;

    if (h_found_flag == FOUND_READY) {
        FoundResult host_result;
        CUDA_CHECK(hipMemcpy(&host_result, d_found_result, sizeof(FoundResult), hipMemcpyDeviceToHost));
        std::cout << "\n======== FOUND MATCH! =================================\n";
        std::cout << "Private Key   : " << CryptoUtils::formatHex256(host_result.scalar_val) << "\n";
        std::cout << "Public Key    : " << CryptoUtils::formatCompressedPubHex(host_result.Rx_val, host_result.Ry_val) << "\n";
        if (verbose) {
            std::cout << "Thread ID     : " << host_result.threadId << "\n";
            std::cout << "Iteration     : " << host_result.iter << "\n";
        }
    } else {
        if (g_sigint) {
            std::cout << "======== INTERRUPTED (Ctrl+C) ==========================\n";
            std::cout << "Search was interrupted by user. Partial progress above.\n";
            exit_code = 130;
        } else if (completed_all) {
            std::cout << "======== KEY NOT FOUND (exhaustive) ===================\n";
            std::cout << "Target hash160 was not found within the specified range.\n";
        } else {
            std::cout << "======== TERMINATED ===================================\n";
            std::cout << "Search terminated due to an error or incomplete range.\n";
        }
    }

    // Cleanup
    CUDA_CHECK(hipFree(d_start_scalars));
    CUDA_CHECK(hipFree(d_counts256));
    CUDA_CHECK(hipFree(d_P));
    CUDA_CHECK(hipFree(d_R));
    CUDA_CHECK(hipFree(d_found_flag));
    CUDA_CHECK(hipFree(d_found_result));
    CUDA_CHECK(hipFree(d_hashes_accum));
    CUDA_CHECK(hipFree(d_any_left));
    CUDA_CHECK(hipFree(d_pre_Gx_local));
    CUDA_CHECK(hipFree(d_pre_Gy_local));
    CUDA_CHECK(hipFree(d_pre_phiGx_local));
    CUDA_CHECK(hipFree(d_pre_phiGy_local));
    if (h_start_scalars) CUDA_CHECK(hipHostFree(h_start_scalars));
    if (h_counts256) CUDA_CHECK(hipHostFree(h_counts256));
    CUDA_CHECK(hipStreamDestroy(streamKernel));

    return exit_code;
}